#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>

#include "GLAD/glad.c"
#include "GLFW/glfw3.h"

#define STB_ONLY_PNG

#define STB_IMAGE_IMPLEMENTATION
#include "STBI/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "STBI/stb_image_write.h"

#include "Debug.h"
#include "Utils.h"
#include "Shader.h"
#include "CopyArt.h"

GLFWwindow* initGL(uint32_t width, uint32_t height) {
    if( !glfwInit() )
    {
        printf("Failed to initialize GLFW\n" );
        return nullptr;
    }

    GLFWwindow* window = glfwCreateWindow(width, height, "Copy Art", NULL, NULL);
    if (window == NULL)
    {
        printf("Failed to create GLFW window\n");
        glfwTerminate();
        return nullptr;
    }
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        printf("Failed to initialize GLAD\n");
        glfwTerminate();
        return nullptr;
    }

    return window;
}

void debug_PrintMem(float* memory, uint32_t size) {

    for(uint32_t i = 0; i < size; i += 6) {
        printf("{%f, %f}, { %f, %f, %f, %f} \n", memory[i], memory[i + 1], memory[i + 2], memory[i + 3], memory[i + 4], memory[i + 5] );
    }

    printf("\n\n\n");
}

__global__ void cuda_fitness(const int N, float* canvasMemory, float* targetMemory, float* storage) {

    __shared__ float cache[1024];

    int t_id = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int cacheId = threadIdx.x;


    float store = 0;
    while(t_id < N) {
        store += abs(canvasMemory[t_id] - targetMemory[t_id]);
        t_id += stride;
    }

    cache[cacheId] = store;

    __syncthreads();

    int i = blockDim.x/2;

    while( i != 0) {
      if(cacheId < i) {
        cache[cacheId] += cache[cacheId + i];
      }
      __syncthreads();
      i /= 2;
    }

    if(cacheId == 0) {
      storage[blockIdx.x] = cache[0];
    }
}

int main() {

    uint32_t width = 180;
    uint32_t height = 180;

    std::string targetPath = "Sunset.png";
    std::string finalImagePath = "out.png";
    std::string frameDirectory = "Frames/";

    uint32_t generationCount = 50000;
    uint32_t populationCount = 100;
    uint32_t triangleCount = 500;

    float mutationRate = .001;
    uint32_t eliteCount = 0;
    float selectionCutoff = .25f;

    GLint drawMode = GL_TRIANGLES;//GL_TRIANGLE_STRIP;
    float pointSize = 5; //IF using GL_POINTS

    bool useCuda = true;
    int N = width*height*3;
    int BLOCKS = 1024;
    int THREADS = 1024;


/* INITILIZATION */

    RNG rng(Timer::now());
    Image target(targetPath);
    Image canvas(width, height);

    std::vector<Specimen> frontPopulation;
    std::vector<Specimen> backPopulation;
    frontPopulation.reserve(populationCount);
    backPopulation.reserve(populationCount);

    std::vector<Specimen>* currentPopulation;
    std::vector<Specimen>* previousPopulation;
    std::vector<Specimen>* tempPopPtr;

    uint32_t floatsPerVertex = 6;
    uint32_t floatPerSpecimen = 3*floatsPerVertex*triangleCount;
    uint32_t drawDataMemSize = populationCount*floatPerSpecimen*sizeof(float);

    float* frontLocalBuffer = (float*)malloc(drawDataMemSize);
    float* backLocalBuffer = (float*)malloc(drawDataMemSize);

    float** currentBuffer;
    float** previousBuffer;
    float** tempBuffer;

/* Statistics Variables */
    float lastScore = 0;
    float curScore = 0;

/*OpenGL Init */
    GLFWwindow* window = initGL(width, height);

    GLuint renderProgram = buildAndLinkShaders(vert_Triangle, frag_Triangle);

    glClearColor(0.5f, 0.5f, 0.5f, 1.0f);
    glEnable(GL_BLEND);
    glPointSize(pointSize);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    Texture targetTex(target, 0);
    Texture offscreenTex(width, height, 1);

    GLuint framebuffer;
    glGenFramebuffers(1, &framebuffer);
    glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
    glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, offscreenTex.handle, 0);

    GLuint vao;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);

    GLuint deviceBuffer;
    glGenBuffers(1, &deviceBuffer);
    glBindBuffer(GL_ARRAY_BUFFER, deviceBuffer);
    glBufferData(GL_ARRAY_BUFFER, drawDataMemSize, NULL, GL_DYNAMIC_READ);

    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, floatsPerVertex*sizeof(float), (void*)0);

    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 4, GL_FLOAT, GL_FALSE, floatsPerVertex*sizeof(float), (void*)(2*sizeof(float)));

/*Initilize CUDA */

    uint32_t imageMemorySize = width*height*3*sizeof(float);
    float* deviceTarget;
    float* deviceCanvas;
    float* deviceStorage;
    float* resultStorage;

    if(useCuda) {
        hipMalloc((void**)&deviceTarget, imageMemorySize);
        hipMalloc((void**)&deviceCanvas, imageMemorySize);
        hipMalloc((void**)&deviceStorage, BLOCKS*sizeof(float));
        resultStorage = (float*)malloc(imageMemorySize);

        hipMemcpy(deviceTarget, (float*)target.pixels, imageMemorySize, hipMemcpyHostToDevice);

        if(!(deviceTarget && deviceCanvas && deviceStorage && resultStorage)) {
            printf("Failed CUDA intilization.\n");
        }
    }

/* Initilize Population */

    for(uint32_t i = 0; i < populationCount; i++) {
        MemoryRange memRange;
        memRange.offset = i*floatPerSpecimen;
        memRange.length = floatPerSpecimen;

        frontPopulation.push_back(Specimen(rng, memRange, frontLocalBuffer));
        backPopulation.push_back(frontPopulation.at(i));

        memcpy(backLocalBuffer, frontLocalBuffer, drawDataMemSize);
    }

    currentPopulation = &frontPopulation;
    currentBuffer = &frontLocalBuffer;

    previousPopulation = &backPopulation;
    previousBuffer = &frontLocalBuffer;//&backLocalBuffer;

/* Begin Simulation */
    for(uint32_t curGen = 0; curGen < generationCount; curGen++) {

        //Upload generation traits
        glBufferSubData(GL_ARRAY_BUFFER, 0, drawDataMemSize, (*currentBuffer));


        for(uint32_t curSpec = 0; curSpec < populationCount; curSpec++) {

            /* Draw the traits */
            glClear(GL_COLOR_BUFFER_BIT);
            glUseProgram(renderProgram);

            uint32_t start = currentPopulation->at(curSpec).memoryRange.offset/floatsPerVertex;
            uint32_t totalOffset = 3*triangleCount;

            glDrawArrays(drawMode, start, totalOffset);

            readFramebuffer(canvas);

            if(useCuda) {
                hipMemcpy(deviceCanvas, canvas.pixels, imageMemorySize, hipMemcpyHostToDevice);
                cuda_fitness <<<BLOCKS, THREADS>>>(N, deviceCanvas, deviceTarget, deviceStorage);
                hipMemcpy(resultStorage, deviceStorage, BLOCKS*sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();

                 for(int i = 0; i < BLOCKS; i++) {
                    currentPopulation->at(curSpec).score +=  resultStorage[i];
                 }
            }
            else {

                currentPopulation->at(curSpec).score = fitness(canvas, target);

            }
        }

        std::sort(currentPopulation->begin(), currentPopulation->end());

        lastScore = curScore;
        curScore = currentPopulation->at(0).score;

        float improvement = (lastScore-curScore)/lastScore*100;
        printf("%d, %f, %f\n", curGen, curScore, improvement);

        /* Swap Population Pointers to build current population */
        tempPopPtr = currentPopulation;
        tempBuffer = currentBuffer;

        currentPopulation = previousPopulation;
        currentBuffer = previousBuffer;

        previousPopulation = tempPopPtr;
        previousBuffer = tempBuffer;

        for(uint32_t curElite = 0; curElite < eliteCount; curElite++) {
            currentPopulation->at(curElite) = previousPopulation->at(curElite);
        }

        /* Selection + Crossover */
        for(uint32_t curSpec = eliteCount; curSpec < populationCount; curSpec++) {

            uint32_t indexA = rng.runifInt(0, populationCount*selectionCutoff - 1);
            uint32_t indexB = rng.runifInt(0, populationCount*selectionCutoff - 1);

            while(indexA == indexB) {
                indexB = rng.runifInt(0, populationCount*selectionCutoff - 1);
            }

            Specimen& specA = previousPopulation->at(indexA);
            Specimen& specB = previousPopulation->at(indexB);
            Specimen& inherit = currentPopulation->at(curSpec);

            currentPopulation->at(curSpec) = Specimen(specA, specB, inherit, rng, mutationRate, (*currentBuffer));


        }

        glBindFramebuffer(GL_READ_FRAMEBUFFER, framebuffer);
        glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
        glBlitFramebuffer(0, 0, width, height, 0, 0, width, height, GL_COLOR_BUFFER_BIT, GL_LINEAR);

        glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);

        glfwPollEvents();
        glfwSwapBuffers(window);

        if(glfwWindowShouldClose(window)) {
            break;
        }
    }


    printf("Completed Simulation.\n");

    while(!glfwWindowShouldClose(window)) {

        glfwPollEvents();
        glfwSwapBuffers(window);
    }


    readFramebufferUByte(canvas);
    canvas.save(finalImagePath);

    glfwTerminate();
    free(frontLocalBuffer);
    free(backLocalBuffer);
    return 0;
}
